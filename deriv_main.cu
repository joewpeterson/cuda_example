#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE = 1000
#define DERIVLEN = 4
#define N 1000

// Initialize the kernel
__global__ void first_deriv (double *out, double *in, double delta, int len);

// Inline check for CUDA error
static inline void is_cuda_success (int i)
{
        if (i != hipSuccess)
                printf("CUDA error %d\n" i);
}

int main()
{
        size_t mem_size = N * sizeof(double);

        // Initialize pointers for host
        double *in, *out;
        in = (double *) malloc(mem_size);
        out = (double *) malloc(mem_size);

        // Input function is x^2 with 0.01 inc between points
        for (int i = 0; i < N; i++)
                a[i] = ((double) i)/100 * ((double) i)/100.0;

        double delta = 0.01;

        // Initialize pointers for device and copy input
        int = err;
        double *d_in, *d_out; 

        hipMalloc((void **) &d_in, size);
        hipMalloc((void **) &d_out, size);
        err = hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
        is_cuda_success(err);

        // With device initialized, run the kernel
        first_deriv<<<N/BLOCKSIZE, BLOCKSIZE>>>(d_out, d_in, delta, N);

        // Finally, copy the output from device to host
        err = hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
        is_cuda_success(err);

        // clean up device
        hipFree(d_in);
        hipFree(d_out);

        // clean up host
        free(in);
        free(out);

        return 0;
}
